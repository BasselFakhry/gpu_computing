#include "hip/hip_runtime.h"

#include "common.h"

#include "timer.h"

__global__ void vecMax_kernel(double* a, double* b, double* c, unsigned int M) {

    // TODO
    int i=blockDim.x*blockIdx.x+threadIdx.x;
    if(i<M){
        if(a[i]>b[i])
            c[i]=a[i];
        else 
            c[i]=b[i];
    }

}

void vecMax_gpu(double* a, double* b, double* c, unsigned int M) {

    Timer timer;

    // Allocate GPU memory
    
    startTime(&timer);
    double *a_d,*b_d,*c_d;
    hipMalloc((void**)&a_d,sizeof(double)*M);
    hipMalloc((void**)&b_d,sizeof(double)*M);
    hipMalloc((void**)&c_d,sizeof(double)*M);
    // TODO





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(a_d,a,sizeof(double)*M,hipMemcpyHostToDevice);
    hipMemcpy(b_d,b,sizeof(double)*M,hipMemcpyHostToDevice);
    
    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);
    const unsigned int nb_threads_per_block=512;
    const unsigned int nb_blocks=(M+nb_threads_per_block-1)/nb_threads_per_block;
    vecMax_kernel<<<nb_blocks,nb_threads_per_block>>>(a_d,b_d,c_d,M);

    // TODO




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time", GREEN);

    // Copy data from GPU
    startTime(&timer);
    hipMemcpy(c,c_d,sizeof(double)*M,hipMemcpyDeviceToHost);
    // TODO


    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    // TODO



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

